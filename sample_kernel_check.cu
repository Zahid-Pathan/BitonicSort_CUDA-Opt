#include <iostream>
#include <hip/hip_runtime.h>

__global__ void hello_kernel() {
    printf("Hello from CUDA!\n");
}

int main() {
    // Launch the kernel
    hello_kernel<<<1, 1>>>();

    // Check for errors during kernel launch
    hipError_t launch_error = hipGetLastError();
    if (launch_error != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(launch_error) << std::endl;
        return 1;
    }
    std::cout << "Kernel launched." << std::endl;

    // Synchronize and check for errors during kernel execution
    hipError_t sync_error = hipDeviceSynchronize();
    if (sync_error != hipSuccess) {
        std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(sync_error) << std::endl;
        return 1;
    }
    std::cout << "Device synchronized." << std::endl;

    return 0;
}